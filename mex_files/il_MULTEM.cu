#include "hip/hip_runtime.h"
/*
 * This file is part of MULTEM.
 * Copyright 2015 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http:// www.gnu.org/licenses/>.
 */

#include <algorithm>

#include "math.cuh"
#include "types.cuh"
#include "matlab_types.cuh"
#include "traits.cuh"
#include "host_device_functions.cuh"
#include "input_multislice.cuh"
#include "output_multislice.hpp"
#include "atom_data.hpp"
#include "multislice.cuh"

#include <mex.h>
#include "matlab_mex.cuh"

using multem::rmatrix_r;
using multem::rmatrix_c;

template<class TInput_Multislice>
void read_input_multislice(const mxArray *mx_input_multislice, TInput_Multislice &input_multislice, bool full =true)
{
	using value_type_r = multem::Value_type<TInput_Multislice>;

	input_multislice.precision = mx_get_scalar_field<multem::ePrecision>(mx_input_multislice, "precision");
	input_multislice.device = mx_get_scalar_field<multem::eDevice>(mx_input_multislice, "device"); 
	input_multislice.cpu_ncores = mx_get_scalar_field<int>(mx_input_multislice, "cpu_ncores"); 
	input_multislice.cpu_nthread = mx_get_scalar_field<int>(mx_input_multislice, "cpu_nthread"); 
	input_multislice.gpu_device = mx_get_scalar_field<int>(mx_input_multislice, "gpu_device"); 
	input_multislice.gpu_nstream = mx_get_scalar_field<int>(mx_input_multislice, "gpu_nstream"); 

	input_multislice.simulation_type = mx_get_scalar_field<multem::eSimulation_Type>(mx_input_multislice, "simulation_type"); 
	input_multislice.phonon_model = mx_get_scalar_field<multem::ePhonon_Model>(mx_input_multislice, "phonon_model"); 
	input_multislice.interaction_model = mx_get_scalar_field<multem::eElec_Spec_Int_Model>(mx_input_multislice, "interaction_model");
	input_multislice.potential_slicing = mx_get_scalar_field<multem::ePotential_Slicing>(mx_input_multislice, "potential_slicing");
	input_multislice.potential_type = mx_get_scalar_field<multem::ePotential_Type>(mx_input_multislice, "potential_type");

	input_multislice.fp_dim.set(mx_get_scalar_field<int>(mx_input_multislice, "fp_dim"));
	input_multislice.fp_seed = mx_get_scalar_field<int>(mx_input_multislice, "fp_seed");
	input_multislice.fp_single_conf = mx_get_scalar_field<bool>(mx_input_multislice, "fp_single_conf");
	input_multislice.fp_nconf = mx_get_scalar_field<int>(mx_input_multislice, "fp_nconf");

	input_multislice.tm_active = mx_get_scalar_field<bool>(mx_input_multislice, "tm_active");
	input_multislice.tm_theta = mx_get_scalar_field<value_type_r>(mx_input_multislice, "tm_theta")*multem::c_deg_2_rad;
	input_multislice.tm_u0 = mx_get_r3d_field<value_type_r>(mx_input_multislice, "tm_u0");
	input_multislice.tm_rot_point_type = mx_get_scalar_field<multem::eRot_Point_Type>(mx_input_multislice, "tm_rot_point_type");
	input_multislice.tm_p0 = mx_get_r3d_field<value_type_r>(mx_input_multislice, "tm_p0");

	input_multislice.microscope_effect = mx_get_scalar_field<multem::eMicroscope_Effect>(mx_input_multislice, "microscope_effect");
	input_multislice.spatial_temporal_effect = mx_get_scalar_field<multem::eSpatial_Temporal_Effect>(mx_input_multislice, "spatial_temporal_effect");

	input_multislice.thickness_type = mx_get_scalar_field<multem::eThickness_Type>(mx_input_multislice, "thickness_type");
	if(!input_multislice.is_whole_specimen() && full)
	{
		auto thickness = mx_get_matrix_field<rmatrix_r>(mx_input_multislice, "thickness");
		input_multislice.thickness.resize(thickness.m_size);
		std::copy(thickness.real, thickness.real + thickness.m_size, input_multislice.thickness.begin());
	}

	input_multislice.operation_mode = mx_get_scalar_field<multem::eOperation_Mode>(mx_input_multislice, "operation_mode");
	input_multislice.coherent_contribution = mx_get_scalar_field<bool>(mx_input_multislice, "coherent_contribution");

	input_multislice.E_0 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "E_0");
	input_multislice.theta = mx_get_scalar_field<value_type_r>(mx_input_multislice, "theta")*multem::c_deg_2_rad;
	input_multislice.phi = mx_get_scalar_field<value_type_r>(mx_input_multislice, "phi")*multem::c_deg_2_rad;

	bool bwl = mx_get_scalar_field<bool>(mx_input_multislice, "bwl");
	bool pbc_xy = true;

	auto nx = mx_get_scalar_field<int>(mx_input_multislice, "nx");
	auto ny = mx_get_scalar_field<int>(mx_input_multislice, "ny");
	auto lx = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lx");
	auto ly = mx_get_scalar_field<value_type_r>(mx_input_multislice, "ly");
	auto dz = mx_get_scalar_field<value_type_r>(mx_input_multislice, "dz"); 				

	auto atoms = mx_get_matrix_field<rmatrix_r>(mx_input_multislice, "atoms");
	if(full)
	{
		input_multislice.atoms.set_Atoms(atoms.rows, atoms.cols, atoms.real, lx, ly);
	}
	input_multislice.grid.set_input_data(nx, ny, lx, ly, dz, bwl, pbc_xy);

	/****************************** Incident wave ********************************/
	auto iw_type = mx_get_scalar_field<multem::eIncident_Wave_Type>(mx_input_multislice, "iw_type");
	input_multislice.set_incident_wave_type(iw_type);

	if(input_multislice.is_user_define_wave() && full)
	{
		auto iw_psi = mx_get_matrix_field<rmatrix_c>(mx_input_multislice, "iw_psi");
		multem::Stream<multem::e_host> stream(input_multislice.cpu_nthread);
		multem::assign(stream, iw_psi, input_multislice.iw_psi);
	}
	input_multislice.iw_x = mx_get_scalar_field<value_type_r>(mx_input_multislice, "iw_x");
	input_multislice.iw_y = mx_get_scalar_field<value_type_r>(mx_input_multislice, "iw_y");

	/****************************** Condenser lens ********************************/
	input_multislice.cond_lens.m = mx_get_scalar_field<int>(mx_input_multislice, "cond_lens_m"); 											// momentum of the vortex
	input_multislice.cond_lens.f = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_f"); 									// defocus(Angstrom)
	input_multislice.cond_lens.Cs3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_Cs3")*multem::c_mm_2_Ags; 			// third order spherical aberration(mm-->Angstrom)
	input_multislice.cond_lens.Cs5 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_Cs5")*multem::c_mm_2_Ags; 			// fifth order aberration(mm-->Angstrom)
	input_multislice.cond_lens.mfa2 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_mfa2"); 							// magnitude 2-fold astigmatism(Angstrom)
	input_multislice.cond_lens.afa2 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_afa2")*multem::c_deg_2_rad; 		// angle 2-fold astigmatism(degrees-->rad)
	input_multislice.cond_lens.mfa3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_mfa3"); 							// magnitude 3-fold astigmatism(Angstrom)
	input_multislice.cond_lens.afa3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_afa3")*multem::c_deg_2_rad; 		// angle 3-fold astigmatism(degrees-->rad)
	input_multislice.cond_lens.inner_aper_ang = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_inner_aper_ang")*multem::c_mrad_2_rad; 		// inner aperture(mrad-->rad)
	input_multislice.cond_lens.outer_aper_ang = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_outer_aper_ang")*multem::c_mrad_2_rad; 		// outer aperture(mrad-->rad)
	input_multislice.cond_lens.sf = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_sf"); 								// defocus spread(Angstrom)
	input_multislice.cond_lens.nsf = mx_get_scalar_field<int>(mx_input_multislice, "cond_lens_nsf"); 										// Number of integration steps for the defocus Spread
	input_multislice.cond_lens.beta = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_beta")*multem::c_mrad_2_rad; 		// divergence semi-angle(mrad-->rad)
	input_multislice.cond_lens.nbeta = mx_get_scalar_field<int>(mx_input_multislice, "cond_lens_nbeta");									// Number of integration steps for the divergence semi-angle
	input_multislice.cond_lens.zero_defocus_type = mx_get_scalar_field<multem::eZero_Defocus_Type>(mx_input_multislice, "cond_lens_zero_defocus_type");
	input_multislice.cond_lens.zero_defocus_plane = mx_get_scalar_field<value_type_r>(mx_input_multislice, "cond_lens_zero_defocus_plane");	
	input_multislice.cond_lens.set_input_data(input_multislice.E_0, input_multislice.grid);

	//input_multislice.cdl_var_type = mx_get_scalar_field<multem::eLens_Var_Type>(mx_input_multislice, "cdl_var_type");
	//if(!input_multislice.is_whole_specimen() && full)
	//{
	//	auto thickness = mx_get_matrix_field<rmatrix_r>(mx_input_multislice, "thickness");
	//	input_multislice.thickness.resize(thickness.m_size);
	//	std::copy(thickness.real, thickness.real + thickness.m_size, input_multislice.thickness.begin());
	//}

	/****************************** Objective lens ********************************/
	input_multislice.obj_lens.m = mx_get_scalar_field<int>(mx_input_multislice, "obj_lens_m"); 												// momentum of the vortex
	input_multislice.obj_lens.f = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_f"); 									// defocus(Angstrom)
	input_multislice.obj_lens.Cs3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_Cs3")*multem::c_mm_2_Ags; 				// third order spherical aberration(mm-->Angstrom)
	input_multislice.obj_lens.Cs5 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_Cs5")*multem::c_mm_2_Ags; 				// fifth order aberration(mm-->Angstrom)
	input_multislice.obj_lens.mfa2 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_mfa2"); 								// magnitude 2-fold astigmatism(Angstrom)
	input_multislice.obj_lens.afa2 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_afa2")*multem::c_deg_2_rad; 			// angle 2-fold astigmatism(degrees-->rad)
	input_multislice.obj_lens.mfa3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_mfa3"); 								// magnitude 3-fold astigmatism(Angstrom)
	input_multislice.obj_lens.afa3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_afa3")*multem::c_deg_2_rad; 			// angle 3-fold astigmatism(degrees-->rad)
	input_multislice.obj_lens.inner_aper_ang = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_inner_aper_ang")*multem::c_mrad_2_rad; 		// inner aperture(mrad-->rad)
	input_multislice.obj_lens.outer_aper_ang = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_outer_aper_ang")*multem::c_mrad_2_rad; 		// outer aperture(mrad-->rad)
	input_multislice.obj_lens.sf = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_sf"); 									// defocus spread(Angstrom)
	input_multislice.obj_lens.nsf = mx_get_scalar_field<int>(mx_input_multislice, "obj_lens_nsf"); 											// Number of integration steps for the defocus Spread
	input_multislice.obj_lens.beta = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_beta")*multem::c_mrad_2_rad; 			// divergence semi-angle(mrad-->rad)
	input_multislice.obj_lens.nbeta = mx_get_scalar_field<int>(mx_input_multislice, "obj_lens_nbeta"); 										// Number of integration steps for the divergence semi-angle
	input_multislice.obj_lens.zero_defocus_type = mx_get_scalar_field<multem::eZero_Defocus_Type>(mx_input_multislice, "obj_lens_zero_defocus_type");
	input_multislice.obj_lens.zero_defocus_plane = mx_get_scalar_field<value_type_r>(mx_input_multislice, "obj_lens_zero_defocus_plane");		
	input_multislice.obj_lens.set_input_data(input_multislice.E_0, input_multislice.grid);

	if(input_multislice.is_scanning())
	{
		input_multislice.scanning.type = mx_get_scalar_field<multem::eScanning_Type>(mx_input_multislice, "scanning_type");
		input_multislice.scanning.pbc = mx_get_scalar_field<bool>(mx_input_multislice, "scanning_periodic");
		input_multislice.scanning.ns = mx_get_scalar_field<int>(mx_input_multislice, "scanning_ns");
		input_multislice.scanning.x0 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "scanning_x0");
		input_multislice.scanning.y0 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "scanning_y0");
		input_multislice.scanning.xe = mx_get_scalar_field<value_type_r>(mx_input_multislice, "scanning_xe");
		input_multislice.scanning.ye = mx_get_scalar_field<value_type_r>(mx_input_multislice, "scanning_ye");
		input_multislice.scanning.set_grid();
	}

	if(input_multislice.is_STEM())
	{
		value_type_r lambda = multem::get_lambda(input_multislice.E_0);
		mxArray *mx_detector = mxGetField(mx_input_multislice, 0, "detector");
		input_multislice.detector.type = mx_get_scalar_field<multem::eDetector_Type>(mx_detector, "type");

		switch (input_multislice.detector.type)
		{
		case multem::eDT_Circular:
			{
				mx_detector = mxGetField(mx_detector, 0, "cir");
				int ndetector = mxGetN(mx_detector);
				if(ndetector>0)
				{
					input_multislice.detector.resize(ndetector);
					for(auto i = 0; i<input_multislice.detector.size(); i++)
					{
						auto inner_ang = mx_get_scalar_field<value_type_r>(mx_detector, i, "inner_ang")*multem::c_mrad_2_rad;
						input_multislice.detector.g_inner[i] = sin(inner_ang)/lambda;
						auto outer_ang = mx_get_scalar_field<value_type_r>(mx_detector, i, "outer_ang")*multem::c_mrad_2_rad;
						input_multislice.detector.g_outer[i] = sin(outer_ang)/lambda;
					}
				}
			}
			break;
		case multem::eDT_Radial:
			{
				mx_detector = mxGetField(mx_detector, 0, "radial");
				int ndetector = mxGetN(mx_detector);
				if(ndetector>0)
				{
					multem::Stream<multem::e_host> stream(input_multislice.cpu_nthread);
					input_multislice.detector.resize(ndetector);
					for(auto i = 0; i<input_multislice.detector.size(); i++)
					{
						// auto x = mx_get_matrix_field<rmatrix_r>(mx_detector, i, "x");
						// multem::assign(x, input_multislice.detector.x[i]);
						// multem::scale(input_multislice.detector.x[i], 1.0/lambda);

						auto fx = mx_get_matrix_field<rmatrix_r>(mx_detector, i, "fx");
						multem::assign(stream, fx, input_multislice.detector.fx[i]);
					}
				}
			}
			break;
		case multem::eDT_Matrix:
			{
				mx_detector = mxGetField(mx_detector, 0, "matrix");
				int ndetector = mxGetN(mx_detector);
				if(ndetector>0)
				{
					multem::Stream<multem::e_host> stream(input_multislice.cpu_nthread);
					input_multislice.detector.resize(ndetector);
					for(auto i = 0; i<input_multislice.detector.size(); i++)
					{
						// auto R = mx_get_matrix_field<rmatrix_r>(mx_detector, i, "x");
						// multem::assign(R, input_multislice.detector.R[i]);
						// multem::scale(input_multislice.detector.R[i], 1.0/lambda);
						// multem::fft2_shift(input_multislice.grid, input_multislice.detector.R[i]);

						auto fR = mx_get_matrix_field<rmatrix_r>(mx_detector, i, "fR");
						multem::assign(stream, fR, input_multislice.detector.fR[i]);
					}
				}
			}
			break;
		}
	}
	else if (input_multislice.is_PED())
	{
		input_multislice.theta = mx_get_scalar_field<value_type_r>(mx_input_multislice, "ped_theta")*multem::c_deg_2_rad;
		input_multislice.nrot = mx_get_scalar_field<value_type_r>(mx_input_multislice, "ped_nrot");
	}
	else if (input_multislice.is_HCI())
	{
		input_multislice.theta = mx_get_scalar_field<value_type_r>(mx_input_multislice, "hci_theta")*multem::c_deg_2_rad;
		input_multislice.nrot = mx_get_scalar_field<value_type_r>(mx_input_multislice, "hci_nrot");
	}
	else if (input_multislice.is_EELS())
	{
		multem::eSpace space = multem::eS_Reciprocal;
		value_type_r E_loss = mx_get_scalar_field<value_type_r>(mx_input_multislice, "eels_E_loss")*multem::c_meV_2_keV;
		int m_selection = mx_get_scalar_field<int>(mx_input_multislice, "eels_m_selection");
		value_type_r collection_angle = mx_get_scalar_field<double>(mx_input_multislice, "eels_collection_angle")*multem::c_mrad_2_rad;
		multem::eChannelling_Type channelling_type = mx_get_scalar_field<multem::eChannelling_Type>(mx_input_multislice, "eels_channelling_type");
		int Z = mx_get_scalar_field<int>(mx_input_multislice, "eels_Z");

		input_multislice.eels_fr.set_input_data(space, input_multislice.E_0, E_loss, m_selection, collection_angle, channelling_type, Z);
	}
	else if (input_multislice.is_EFTEM())
	{
		multem::eSpace space = multem::eS_Real;
		value_type_r E_loss = mx_get_scalar_field<value_type_r>(mx_input_multislice, "eftem_E_loss")*multem::c_meV_2_keV;
		int m_selection = mx_get_scalar_field<int>(mx_input_multislice, "eftem_m_selection");
		value_type_r collection_angle = mx_get_scalar_field<double>(mx_input_multislice, "eftem_collection_angle")*multem::c_mrad_2_rad;
		multem::eChannelling_Type channelling_type = mx_get_scalar_field<multem::eChannelling_Type>(mx_input_multislice, "eftem_channelling_type");
		int Z = mx_get_scalar_field<int>(mx_input_multislice, "eftem_Z");

		input_multislice.eels_fr.set_input_data(space, input_multislice.E_0, E_loss, m_selection, collection_angle, channelling_type, Z);
	}

	input_multislice.validate_parameters();
 }

void set_output_multislice(const mxArray *mx_input_multislice, mxArray *&mx_output_multislice, multem::Output_Multislice_Matlab &output_multislice)
{
	multem::Input_Multislice<double> input_multislice;
	read_input_multislice(mx_input_multislice, input_multislice);
	output_multislice.set_input_data(&input_multislice);

	const char *field_names_output_multislice[] = {"dx", "dy", "x", "y", "thickness", "data"};
	int number_of_fields_output_multislice = 6;
	mwSize dims_output_multislice[2] = {1, 1};

	mx_output_multislice = mxCreateStructArray(2, dims_output_multislice, number_of_fields_output_multislice, field_names_output_multislice);

	mx_create_set_scalar_field<rmatrix_r>(mx_output_multislice, "dx", output_multislice.dx);
	mx_create_set_scalar_field<rmatrix_r>(mx_output_multislice, "dy", output_multislice.dy);
	mx_create_set_matrix_field<rmatrix_r>(mx_output_multislice, "x", 1, output_multislice.x.size(), output_multislice.x.data());
	mx_create_set_matrix_field<rmatrix_r>(mx_output_multislice, "y", 1, output_multislice.y.size(), output_multislice.y.data());
	mx_create_set_matrix_field<rmatrix_r>(mx_output_multislice, "thickness", 1, output_multislice.thickness.size(), output_multislice.thickness.data());

	if(output_multislice.is_STEM() || output_multislice.is_EELS())
	{
		mxArray *mx_field_data;
		const char *field_names_data_full[] = {"image_tot", "image_coh"};
		const char *field_names_data_partial[] = {"image_tot"};
		const char **field_names_data = (output_multislice.coherent_contribution)?field_names_data_full:field_names_data_partial;
		int number_of_fields_data = (output_multislice.coherent_contribution)?2:1;
		mwSize dims_data[2] = {1, output_multislice.thickness.size()};

		mx_field_data = mxCreateStructArray(2, dims_data, number_of_fields_data, field_names_data);
		mxSetField(mx_output_multislice, 0, "data", mx_field_data);

		mxArray *mx_field_detector_tot;
		mxArray *mx_field_detector_coh;
		const char *field_names_detector[] = {"image"};
		int number_of_fields_detector = 1;
		// mwSize dims_detector[2] = {1, output_multislice.ndetector};
		mwSize dims_detector[2];
		dims_detector[0] = 1;
		dims_detector[1] = output_multislice.ndetector;

		for(auto ithk = 0; ithk<output_multislice.thickness.size(); ithk++)
		{
			mx_field_detector_tot = mxCreateStructArray(2, dims_detector, number_of_fields_detector, field_names_detector);
			mxSetField(mx_field_data, ithk, "image_tot", mx_field_detector_tot);
			if(output_multislice.coherent_contribution)
			{
				mx_field_detector_coh = mxCreateStructArray(2, dims_detector, number_of_fields_detector, field_names_detector);
				mxSetField(mx_field_data, ithk, "image_coh", mx_field_detector_coh);
			}

			for(auto iDet = 0; iDet<output_multislice.ndetector; iDet++)
			{
				output_multislice.image_tot[ithk].image[iDet] = mx_create_matrix_field<rmatrix_r>(mx_field_detector_tot, iDet, "image", output_multislice.ny, output_multislice.nx);
				if(output_multislice.coherent_contribution)
				{
					output_multislice.image_coh[ithk].image[iDet] = mx_create_matrix_field<rmatrix_r>(mx_field_detector_coh, iDet, "image", output_multislice.ny, output_multislice.nx);
				}
			}
		}
	}
	else if(output_multislice.is_EWFS_EWRS())
	{
		mxArray *mx_field_data;
		const char *field_names_data_full[] = {"m2psi_tot", "psi_coh"};
		const char *field_names_data_partial[] = {"psi_coh"};
		const char **field_names_data = (!output_multislice.is_EWFS_EWRS_SC())?field_names_data_full:field_names_data_partial;
		int number_of_fields_data = (!output_multislice.is_EWFS_EWRS_SC())?2:1;
		mwSize dims_data[2] = {1, output_multislice.thickness.size()};

		mx_field_data = mxCreateStructArray(2, dims_data, number_of_fields_data, field_names_data);
		mxSetField(mx_output_multislice, 0, "data", mx_field_data);

		for(auto ithk = 0; ithk<output_multislice.thickness.size(); ithk++)
		{
			if(!output_multislice.is_EWFS_EWRS_SC())
			{
				output_multislice.m2psi_tot[ithk] = mx_create_matrix_field<rmatrix_r>(mx_field_data, ithk, "m2psi_tot", output_multislice.ny, output_multislice.nx);
			}
			output_multislice.psi_coh[ithk] = mx_create_matrix_field<rmatrix_c>(mx_field_data, ithk, "psi_coh", output_multislice.ny, output_multislice.nx);
		}
	}
	else
	{
		mxArray *mx_field_data;
		const char *field_names_data_full[] = {"m2psi_tot", "m2psi_coh"};
		const char *field_names_data_partial[] = {"m2psi_tot"};
		const char **field_names_data = (output_multislice.coherent_contribution)?field_names_data_full:field_names_data_partial;
		int number_of_fields_data = (output_multislice.coherent_contribution)?2:1;
		mwSize dims_data[2] = {1, output_multislice.thickness.size()};

		mx_field_data = mxCreateStructArray(2, dims_data, number_of_fields_data, field_names_data);
		mxSetField(mx_output_multislice, 0, "data", mx_field_data);

		for(auto ithk = 0; ithk<output_multislice.thickness.size(); ithk++)
		{
			output_multislice.m2psi_tot[ithk] = mx_create_matrix_field<rmatrix_r>(mx_field_data, ithk, "m2psi_tot", output_multislice.ny, output_multislice.nx);
			if(output_multislice.coherent_contribution)
			{
				output_multislice.m2psi_coh[ithk] = mx_create_matrix_field<rmatrix_r>(mx_field_data, ithk, "m2psi_coh", output_multislice.ny, output_multislice.nx);
			}
		}
	}
}

template<class T, multem::eDevice dev>
void get_multislice(const mxArray *mxB, multem::Output_Multislice_Matlab &output_multislice)
{
	multem::Input_Multislice<T> input_multislice;
	read_input_multislice(mxB, input_multislice);

	multem::Multislice<T, dev> multislice;
	multislice.set_input_data(&input_multislice);

	multislice.run(output_multislice);

	multislice.cleanup();
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
	multem::Output_Multislice_Matlab output_multislice;
	set_output_multislice(prhs[0], plhs[0], output_multislice);

	if(output_multislice.is_float_host())
	{
		get_multislice<float, multem::e_host>(prhs[0], output_multislice);
	}
	else if(output_multislice.is_double_host())
	{
		get_multislice<double, multem::e_host>(prhs[0], output_multislice);
	}
	if(output_multislice.is_float_device())
	{
		get_multislice<float, multem::e_device>(prhs[0], output_multislice);
	}
	else if(output_multislice.is_double_device())
	{
		get_multislice<double, multem::e_device>(prhs[0], output_multislice);
	}
}