#include "hip/hip_runtime.h"
/*
 * This file is part of MULTEM.
 * Copyright 2015 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http:// www.gnu.org/licenses/>.
 */

#include <algorithm>

#include "math.cuh"
#include "types.cuh"
#include "matlab_types.cuh"
#include "traits.cuh"
#include "stream.cuh"
#include "atomic_data.hpp"
#include "host_device_functions.cuh"
#include "input_multislice.cuh"
#include "output_multislice.hpp"
#include "atom_data.hpp"
#include "atomic_cross_section.cuh"

#include <mex.h>
#include "matlab_mex.cuh"

using multem::Vector;
using multem::rmatrix_r;
using multem::rmatrix_c;
using multem::e_host;

class Output_Cross_section
{
	public:
		Output_Cross_section(): nr(0){}

		template<class TInput_Multislice>
		void set_input_data(TInput_Multislice *input_multislice)
		{ 
			nr = input_multislice->scanning.ns;

			stream.resize(input_multislice->cpu_nthread);
		}

		void init()
		{ 
			multem::fill(stream, r, 0);
			multem::fill(stream, fr, 0);
		}

		int nr;
		rmatrix_r r;
		rmatrix_r fr;

		multem::Stream<e_host> stream;
};

template<class TInput_Multislice>
void read_input_multislice(const mxArray *mx_input_multislice, TInput_Multislice &input_multislice)
{
	using value_type_r = multem::Value_type<TInput_Multislice>;

	input_multislice.precision = multem::eP_float;
	input_multislice.device = multem::e_device; 
	input_multislice.cpu_ncores = 1; 
	input_multislice.cpu_nthread = 4; 
	input_multislice.gpu_device = 0;
	input_multislice.gpu_nstream = 1;
	input_multislice.set_device();

	input_multislice.simulation_type = multem::eST_STEM ;
	input_multislice.phonon_model = multem::ePM_Still_Atom;
	input_multislice.interaction_model = multem::eESIM_Multislice;
	input_multislice.potential_slicing = multem::ePS_dz_Sub;
	input_multislice.potential_type = multem::ePT_Lobato_0_12;

	input_multislice.zero_defocus_type = multem::eZDT_Last;
	input_multislice.zero_defocus_plane = 0.0;

	input_multislice.E_0 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "E_0");
	int Z = mx_get_scalar_field<int>(mx_input_multislice, "Z");
	double rms3d = mx_get_scalar_field<double>(mx_input_multislice, "rms3d");
	double fwsig = mx_get_scalar_field<double>(mx_input_multislice, "fwhm")*multem::c_fwhm2sigma;

	bool bwl = false;
	bool pbc_xy = false;

	int nx = 1024;
	int ny = 1024;
	double lx = 20;
	double ly = 20;
	double dz = 0.4; 				

	/******************************** set atom *********************************/
	int natoms = 1;
	double atoms[6];
	atoms[0] = Z; 
	atoms[1] = 0.5*lx; 
	atoms[2] = 0.5*ly; 
	atoms[3] = 0.0; 
	atoms[4] = sqrt(rms3d*rms3d+fwsig*fwsig); 
	atoms[5] = 1.0;
	input_multislice.atoms.set_Atoms(natoms, atoms, lx, ly);
	input_multislice.grid.set_input_data(nx, ny, lx, ly, dz, bwl, pbc_xy);

	/******************************* aberrations *********************************/
	input_multislice.lens.m = mx_get_scalar_field<int>(mx_input_multislice, "lens_m"); 											// momentum of the vortex
	input_multislice.lens.f = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_f"); 									// defocus(Angstrom)
	input_multislice.lens.Cs3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_Cs3")*multem::c_mm_2_Ags; 			// spherical aberration(mm-->Angstrom)
	input_multislice.lens.Cs5 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_Cs5")*multem::c_mm_2_Ags; 			// spherical aberration(mm-->Angstrom)
	input_multislice.lens.mfa2 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_mfa2"); 							// magnitude 2-fold astigmatism(Angstrom)
	input_multislice.lens.afa2 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_afa2")*multem::c_deg_2_rad; 		// angle 2-fold astigmatism(degrees-->rad)
	input_multislice.lens.mfa3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_mfa3"); 							// magnitude 3-fold astigmatism(Angstrom)
	input_multislice.lens.afa3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_afa3")*multem::c_deg_2_rad; 		// angle 3-fold astigmatism(degrees-->rad)
	input_multislice.lens.aobjl = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_aobjl")*multem::c_mrad_2_rad; 	// lower objective aperture(mrad-->rad)
	input_multislice.lens.aobju = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_aobju")*multem::c_mrad_2_rad; 	// upper objective aperture(mrad-->rad)
	input_multislice.lens.set_input_data(input_multislice.E_0, input_multislice.grid);

	/********************************* Detectors ********************************/
	value_type_r lambda = multem::get_lambda(input_multislice.E_0);
	mxArray *mx_detector = mxGetField(mx_input_multislice, 0, "detector");
	input_multislice.detector.type = multem::eDT_Circular;
	mx_detector = mxGetField(mx_detector, 0, "cir");
	input_multislice.detector.resize(1);
	input_multislice.detector.g_inner[0] = mx_get_scalar_field<value_type_r>(mx_detector, 0, "ang_inner")*multem::c_mrad_2_rad/lambda;
	input_multislice.detector.g_outer[0] = mx_get_scalar_field<value_type_r>(mx_detector, 0, "ang_outer")*multem::c_mrad_2_rad/lambda;

	/********************************* Scanning ********************************/
	multem::Atom_Cal<double> atom_cal;
	multem::Atomic_Data atomic_data;
	atomic_data.Load_Data(input_multislice.potential_type);
	multem::Atom_Type<double, multem::e_host> atom_type;

	atomic_data.To_atom_type_CPU(Z, multem::c_Vrl, multem::c_nR, 0.0, atom_type);
	atom_cal.Set_Atom_Type(input_multislice.potential_type, &atom_type);
	auto rmax = atom_cal.AtomicRadius_Cutoff(3, 0.005);

	input_multislice.scanning.type = multem::eST_Line;
	input_multislice.scanning.grid_type = multem::eGT_Regular;
	input_multislice.scanning.ns = multem::c_nR;
	input_multislice.scanning.x0 = 0.5*lx;
	input_multislice.scanning.y0 = 0.5*ly;
	input_multislice.scanning.xe = 0.5*lx;
	input_multislice.scanning.ye = 0.5*ly+rmax;
	input_multislice.scanning.set_grid();

	input_multislice.validate_parameters();
 }

template<class TOutput_Cross_Section>
void set_output_cross_section(const mxArray *mx_input_cross_section, mxArray *&mx_output_cross_section, TOutput_Cross_Section &output_cross_section)
{
	multem::Input_Multislice<double> input_multislice;
	read_input_multislice(mx_input_cross_section, input_multislice);
	output_cross_section.set_input_data(&input_multislice);

	const char *field_names_output_multislice[] = {"r", "fr"};
	int number_of_fields_output_multislice = 2;
	mwSize dims_output_multislice[2] = {1, 1};

	mx_output_cross_section = mxCreateStructArray(2, dims_output_multislice, number_of_fields_output_multislice, field_names_output_multislice);
	output_cross_section.r = mx_create_matrix_field<rmatrix_r>(mx_output_cross_section, "r", 1, output_cross_section.nr);
	output_cross_section.fr = mx_create_matrix_field<rmatrix_r>(mx_output_cross_section, "fr", 1, output_cross_section.nr);
}

template<class T, multem::eDevice dev, class TOutput_Cross_Section>
void get_cross_section(const mxArray *mxB, TOutput_Cross_Section &output_cross_section)
{
	/**************************multislice calculation*******************************/
	multem::Input_Multislice<T> input_multislice;
	read_input_multislice(mxB, input_multislice);

	multem::Atomic_Cross_Section<T, dev> atomic_cross_section;
	atomic_cross_section.set_input_data(&input_multislice);

	atomic_cross_section.get(output_cross_section.r, output_cross_section.fr);
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
	Output_Cross_section output_cross_section;
	set_output_cross_section(prhs[0], plhs[0], output_cross_section);

	get_cross_section<float, multem::e_device>(prhs[0], output_cross_section);
}