#include "hip/hip_runtime.h"
/*
 * This file is part of MULTEM.
 * Copyright 2015 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http:// www.gnu.org/licenses/>.
 */

#include <algorithm>

#include "math.cuh"
#include "types.cuh"
#include "matlab_types.cuh"
#include "traits.cuh"

#include "host_device_functions.cuh"
#include "input_multislice.cuh"
#include "output_multislice.hpp"
#include "atom_data.hpp"
#include "multislice.cuh"
#include "atomic_cross_section.cuh"

#include "input_tomography.cuh"
#include "output_tomography.hpp"
#include "tomography.cuh"

#include <mex.h>
#include "matlab_mex.cuh"

using multem::rmatrix_r;
using multem::rmatrix_c;
using multem::e_host;
using multem::e_device;

template<class TInput_Multislice>
void read_input_multislice(const mxArray *mx_input_multislice, TInput_Multislice &input_multislice)
{
	using value_type_r = multem::Value_type<TInput_Multislice>;

	input_multislice.precision = multem::eP_float;
	input_multislice.device = multem::e_device; 
	input_multislice.cpu_ncores = 1; 
	input_multislice.cpu_nthread = 4; 
	input_multislice.gpu_device = 0;
	input_multislice.gpu_nstream = 1;
	input_multislice.set_device();

	input_multislice.simulation_type = multem::eST_STEM ;
	input_multislice.phonon_model = multem::ePM_Still_Atom;
	input_multislice.interaction_model = multem::eESIM_Multislice;
	input_multislice.potential_slicing = multem::ePS_dz_Sub;
	input_multislice.potential_type = multem::ePT_Lobato_0_12;

	input_multislice.zero_defocus_type = multem::eZDT_Last;
	input_multislice.zero_defocus_plane = 0.0;

	input_multislice.E_0 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "E_0");
	int Z = mx_get_scalar_field<int>(mx_input_multislice, "Z");
	double rms3d = mx_get_scalar_field<double>(mx_input_multislice, "rms3d");
	double fwsig = mx_get_scalar_field<double>(mx_input_multislice, "fwhm")*multem::c_fwhm2sigma;

	bool bwl = false;
	bool pbc_xy = true;

	int nx = 1024;
	int ny = 1024;
	double lx = 20;
	double ly = 20;
	double dz = 0.4; 				

	/******************************** set atom *********************************/
	int natoms = 1;
	double atoms[6];
	atoms[0] = Z; 
	atoms[1] = 0.5*lx; 
	atoms[2] = 0.5*ly; 
	atoms[3] = 0.0; 
	atoms[4] = sqrt(rms3d*rms3d+fwsig*fwsig); 
	atoms[5] = 1.0;
	input_multislice.atoms.set_Atoms(natoms, atoms, lx, ly);
	input_multislice.grid.set_input_data(nx, ny, lx, ly, dz, bwl, pbc_xy);

	/******************************* aberrations *********************************/
	input_multislice.lens.m = mx_get_scalar_field<int>(mx_input_multislice, "lens_m"); 											// momentum of the vortex
	input_multislice.lens.f = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_f"); 									// defocus(Angstrom)
	input_multislice.lens.Cs3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_Cs3")*multem::c_mm_2_Ags; 			// spherical aberration(mm-->Angstrom)
	input_multislice.lens.Cs5 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_Cs5")*multem::c_mm_2_Ags; 			// spherical aberration(mm-->Angstrom)
	input_multislice.lens.mfa2 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_mfa2"); 							// magnitude 2-fold astigmatism(Angstrom)
	input_multislice.lens.afa2 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_afa2")*multem::c_deg_2_rad; 		// angle 2-fold astigmatism(degrees-->rad)
	input_multislice.lens.mfa3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_mfa3"); 							// magnitude 3-fold astigmatism(Angstrom)
	input_multislice.lens.afa3 = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_afa3")*multem::c_deg_2_rad; 		// angle 3-fold astigmatism(degrees-->rad)
	input_multislice.lens.aobjl = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_aobjl")*multem::c_mrad_2_rad; 	// lower objective aperture(mrad-->rad)
	input_multislice.lens.aobju = mx_get_scalar_field<value_type_r>(mx_input_multislice, "lens_aobju")*multem::c_mrad_2_rad; 	// upper objective aperture(mrad-->rad)
	input_multislice.lens.set_input_data(input_multislice.E_0, input_multislice.grid);

	/********************************* Detectors ********************************/
	value_type_r lambda = multem::get_lambda(input_multislice.E_0);
	mxArray *mx_detector = mxGetField(mx_input_multislice, 0, "detector");
	input_multislice.detector.type = multem::eDT_Circular;
	mx_detector = mxGetField(mx_detector, 0, "cir");
	input_multislice.detector.resize(1);
	input_multislice.detector.g_inner[0] = mx_get_scalar_field<value_type_r>(mx_detector, 0, "ang_inner")*multem::c_mrad_2_rad/lambda;
	input_multislice.detector.g_outer[0] = mx_get_scalar_field<value_type_r>(mx_detector, 0, "ang_outer")*multem::c_mrad_2_rad/lambda;

	/********************************* Scanning ********************************/
	multem::Atom_Cal<double> atom_cal;
	multem::Atomic_Data atomic_data;
	atomic_data.Load_Data(input_multislice.potential_type);
	multem::Atom_Type<double, multem::e_host> atom_type;

	atomic_data.To_atom_type_CPU(Z, multem::c_Vrl, multem::c_nR, 0.0, atom_type);
	atom_cal.Set_Atom_Type(input_multislice.potential_type, &atom_type);
	auto rmax = atom_cal.AtomicRadius_Cutoff(3, 0.005);

	input_multislice.scanning.type = multem::eST_Line;
	input_multislice.scanning.grid_type = multem::eGT_Regular;
	input_multislice.scanning.ns = multem::c_nR;
	input_multislice.scanning.x0 = 0.5*lx;
	input_multislice.scanning.y0 = 0.5*ly;
	input_multislice.scanning.xe = 0.5*lx;
	input_multislice.scanning.ye = 0.5*ly+rmax;
	input_multislice.scanning.set_grid();

	input_multislice.validate_parameters();
 }

template<class TInput_Tomography>
void read_tomography(const mxArray *mx_input_tomography, TInput_Tomography &input_tomography, bool full =true)
{
	using value_type_r = multem::Value_type<TInput_Tomography>;

	input_tomography.precision = mx_get_scalar_field<multem::ePrecision>(mx_input_tomography, "precision");
	input_tomography.device = mx_get_scalar_field<multem::eDevice>(mx_input_tomography, "device"); 
	input_tomography.cpu_nthread = mx_get_scalar_field<int>(mx_input_tomography, "cpu_nthread"); 
	input_tomography.gpu_device = mx_get_scalar_field<int>(mx_input_tomography, "gpu_device"); 
	input_tomography.gpu_nstream = mx_get_scalar_field<int>(mx_input_tomography, "gpu_nstream"); 
	input_tomography.set_device();

	input_tomography.tm_u0 = mx_get_r3d_field<value_type_r>(mx_input_tomography, "tm_u0");
	input_tomography.tm_p0 = mx_get_r3d_field<value_type_r>(mx_input_tomography, "tm_p0");
	/***************************get cross section*****************************/
	input_tomography.Z = mx_get_scalar_field<int>(mx_input_tomography, "Z");

	if(full)
	{
		multem::Input_Multislice<float, e_device> input_multislice;
		read_input_multislice(mx_input_tomography, input_multislice);

		multem::Atomic_Cross_Section<float, e_device> atomic_cross_section;
		atomic_cross_section.set_input_data(&input_multislice);

		atomic_cross_section.get(input_tomography.r, input_tomography.fr);
	}

	/*************************************************************************/
	auto angle = mx_get_matrix_field<rmatrix_r>(mx_input_tomography, 0, "angle");
	multem::Vector<value_type_r, e_host> angle_host;
	multem::assign(angle, angle_host);
	multem::scale(angle_host, multem::c_deg_2_rad);
	multem::assign(angle_host, input_tomography.angle);

	mxArray *mx_data = mxGetField(mx_input_tomography, 0, "data");	
	if(full)
	{
		int nimage = mxGetN(mx_data)*mxGetM(mx_data);
		input_tomography.image.resize(nimage);
		for(auto i = 0; i<input_tomography.image.size(); i++)
		{
			auto image = mx_get_matrix_field<rmatrix_r>(mx_data, i, "image");
			multem::assign(image, input_tomography.image[i]);
		}
	}

	value_type_r dR = mx_get_scalar_field<value_type_r>(mx_input_tomography, "dR");
	auto image = mx_get_matrix_field<rmatrix_r>(mx_data, 0, "image");
	bool bwl = false;
	bool pbc_xy = false;

	int nx = image.cols;
	int ny = image.rows;
	value_type_r lx = nx*dR;
	value_type_r ly = ny*dR;
	value_type_r dz = 0.5; 

	input_tomography.input_atoms = mx_get_scalar_field<multem::eInput_Atoms>(mx_input_tomography, "input_atoms");

	auto atoms = mx_get_matrix_field<rmatrix_r>(mx_input_tomography, "atoms");
	if((input_tomography.is_input_atoms())&&(atoms.rows>0))
	{
		auto atoms_min = mx_get_matrix_field<rmatrix_r>(mx_input_tomography, "atoms_min");
		auto atoms_max = mx_get_matrix_field<rmatrix_r>(mx_input_tomography, "atoms_max");
		input_tomography.atoms.set_Atoms(atoms.rows, atoms.real, atoms_min.real, atoms_max.real);
	}
	else
	{	
		auto natoms = mx_get_scalar_field<int>(mx_input_tomography, "natoms"); 
		input_tomography.atoms.resize(natoms);
	}
	input_tomography.grid.set_input_data(nx, ny, lx, ly, dz, bwl, pbc_xy);

	input_tomography.r0_min = mx_get_scalar_field<value_type_r>(mx_input_tomography, "r0_min"); 
	input_tomography.rTemp = mx_get_scalar_field<value_type_r>(mx_input_tomography, "rTemp"); 

	input_tomography.validate_parameters();
 }

void set_output_tomography(const mxArray *mx_input_tomography, mxArray *&mx_output_tomography, multem::Output_Tomography_Matlab &output_tomography)
{
	multem::Input_Tomography<double> input_tomography;
	read_tomography(mx_input_tomography, input_tomography, false);
	output_tomography.set_input_data(&input_tomography);

	const char *field_names_output_tomography[] = {"temp", "chi2", "atoms"};
	int number_of_fields_output_tomography = 3;
	mwSize dims_output_tomography[2] = {1, 1};

	mx_output_tomography = mxCreateStructArray(2, dims_output_tomography, number_of_fields_output_tomography, field_names_output_tomography);

	// output_tomography.temp = mx_create_matrix_field<rmatrix_r>(mx_output_tomography, "temp", output_tomography.temp.m_size, 1);
	// output_tomography.chi2 = mx_create_matrix_field<rmatrix_r>(mx_output_tomography, "chi2", output_tomography.chi2.m_size, 1);

	auto atoms = mx_create_matrix_field<rmatrix_r>(mx_output_tomography, "atoms", input_tomography.atoms.size(), 4);

	output_tomography.Z.rows = atoms.rows;
	output_tomography.Z.cols = 1;
	output_tomography.Z.m_size = output_tomography.Z.rows*output_tomography.Z.cols;
	output_tomography.Z.real = atoms.real + 0*atoms.rows;

	output_tomography.x.rows = atoms.rows;
	output_tomography.x.cols = 1;
	output_tomography.x.m_size = output_tomography.x.rows*output_tomography.x.cols;
	output_tomography.x.real = atoms.real + 1*atoms.rows;

	output_tomography.y.rows = atoms.rows;
	output_tomography.y.cols = 1;
	output_tomography.y.m_size = output_tomography.y.rows*output_tomography.y.cols;
	output_tomography.y.real = atoms.real + 2*atoms.rows;

	output_tomography.z.rows = atoms.rows;
	output_tomography.z.cols = 1;
	output_tomography.z.m_size = output_tomography.z.rows*output_tomography.z.cols;
	output_tomography.z.real = atoms.real + 3*atoms.rows;
}

template<class T, multem::eDevice dev>
void il_tomography(const mxArray *mxB, multem::Output_Tomography_Matlab &output_tomography)
{
	multem::Input_Tomography<T> input_tomography;
	read_tomography(mxB, input_tomography);

	multem::Tomography<T, dev> tomography;
	tomography.set_input_data(&input_tomography);

	tomography.run(output_tomography);
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
	multem::Output_Tomography_Matlab output_tomography;
	set_output_tomography(prhs[0], plhs[0], output_tomography);

	if(output_tomography.is_float_host())
	{
		il_tomography<float, multem::e_host>(prhs[0], output_tomography);
	}
	else if(output_tomography.is_double_host())
	{
		il_tomography<double, multem::e_host>(prhs[0], output_tomography);
	}
	if(output_tomography.is_float_device())
	{
		il_tomography<float, multem::e_device>(prhs[0], output_tomography);
	}
	else if(output_tomography.is_double_device())
	{
		il_tomography<double, multem::e_device>(prhs[0], output_tomography);
	}
}