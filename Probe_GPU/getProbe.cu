#include "hip/hip_runtime.h"
/*
 * This file is part of MULTEM.
 * Copyright 2014 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http://www.gnu.org/licenses/>.
 */

#include "hConstTypes.h"
#include "hMT_General_CPU.h"
#include "hMT_General_GPU.h"
#include "hMT_IncidentWave_GPU.h"
#include "hMatlab2Cpp.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include "hipfft/hipfft.h"
#include <mex.h>

class cProbe{
	private:
		double x0;
		double y0;
		cMT_MGP_CPU MT_MGP_CPU;
		sGP GP;
		sLens Lens;
		sComplex Psii;
		double2 *Psi;
		hipfftHandle PlanPsi;
		cMT_IncidentWave_GPU MT_IncidentWave_GPU;
	public:
		void freeMemory();
		cProbe();
		~cProbe();
		void SetInputData(sInProbe &InProbe);
		void getProbe(sComplex &Psih);
};

void cProbe::freeMemory(){
	hipDeviceSynchronize(); // wait to finish the work in the GPU

	f_sGP_Init(GP);
	f_sLens_Init(Lens);

	cudaFreen(Psii.real);
	cudaFreen(Psii.imag);
	cudaFreen(Psi);
	cufftDestroyn(PlanPsi);
}

cProbe::~cProbe(){
	freeMemory();
}

cProbe::cProbe(){
	f_sGP_Init(GP);
	f_sLens_Init(Lens);
	Psii.real = 0;
	Psii.imag = 0;
	Psi = 0;
	PlanPsi = 0;
}

void cProbe::SetInputData(sInProbe &InProbe){
	freeMemory();

	MT_MGP_CPU.gpu = InProbe.gpu;
	MT_MGP_CPU.E0 = InProbe.E0;	
	MT_MGP_CPU.theta = InProbe.theta;	
	MT_MGP_CPU.phi = InProbe.phi;
	MT_MGP_CPU.lx = InProbe.lx;
	MT_MGP_CPU.ly = InProbe.ly;
	MT_MGP_CPU.nx = InProbe.nx;
	MT_MGP_CPU.ny = InProbe.ny;
	MT_MGP_CPU.BWL = true;

	hipSetDevice(MT_MGP_CPU.gpu);

	f_sGP_Cal(MT_MGP_CPU.nx, MT_MGP_CPU.ny, MT_MGP_CPU.lx, MT_MGP_CPU.ly, MT_MGP_CPU.dz, MT_MGP_CPU.PBC_xy, MT_MGP_CPU.BWL, GP);

	Lens.m = InProbe.m;
	Lens.f = InProbe.f;
	Lens.Cs3 = InProbe.Cs3;
	Lens.Cs5 = InProbe.Cs5;
	Lens.mfa2 = InProbe.mfa2;
	Lens.afa2 = InProbe.afa2;
	Lens.mfa3 = InProbe.mfa3;
	Lens.afa3 = InProbe.afa3;
	Lens.aobjl = InProbe.aobjl;
	Lens.aobju = InProbe.aobju;
	Lens.sf = InProbe.sf;
	Lens.nsf = InProbe.nsf;
	Lens.beta = InProbe.beta;
	Lens.nbeta = InProbe.nbeta;
	f_sLens_Cal(MT_MGP_CPU.E0, GP, Lens);

	double gxu = sin(MT_MGP_CPU.theta)*cos(MT_MGP_CPU.phi)/Lens.lambda;
	double gyu = sin(MT_MGP_CPU.theta)*sin(MT_MGP_CPU.phi)/Lens.lambda;

	x0 = InProbe.x0;
	y0 = InProbe.y0;

	hipMalloc((void**)&(Psii.real), GP.nxy*cSizeofRD);
	hipMalloc((void**)&(Psii.imag), GP.nxy*cSizeofRD);
	hipMalloc((void**)&Psi, GP.nxy*cSizeofCD);
	hipfftPlan2d(&PlanPsi, GP.nx, GP.ny, HIPFFT_Z2Z);

	MT_IncidentWave_GPU.SetInputData(GP, Lens, PlanPsi);
}

void cProbe::getProbe(sComplex &Psih){
	MT_IncidentWave_GPU.Psi0(x0, y0, Psi);
	// fft2shift 
	f_fft2Shift_MC(GP, Psi);
	/*********************copy data to host************************/
	f_Copy_MCd(GP, Psi, Psii.real, Psii.imag, Psih);
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
	sInProbe InProbe;
	sComplex Psih;
	cProbe Probe;

	f_Matlab2InProbe(prhs[0], InProbe);
	/************************Output data**************************/
	plhs[0] = mxCreateDoubleMatrix(InProbe.ny, InProbe.nx, mxCOMPLEX);
	Psih.real = mxGetPr(plhs[0]);
	Psih.imag = mxGetPi(plhs[0]);

	Probe.SetInputData(InProbe);
	Probe.getProbe(Psih);
}