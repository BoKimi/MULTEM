/*
 * This file is part of MULTEM.
 * Copyright 2014 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http://www.gnu.org/licenses/>.
 */

#include "hConstTypes.h"
#include "hTEMIm.h"
#include "hMT_General_CPU.h"
#include "hMT_General_GPU.h"
#include "hMT_MicroscopeEffects_GPU.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hipfft/hipfft.h>

cTEMIm::cTEMIm()
{
	GPU_Device = 0;
	MEffect = 0;
	Psirh = 0;
	Psiih = 0;
	E0 = 0;
	lx = 0;
	ly = 0;
	nx = 0;
	ny = 0;
	lambda = 0;

	fPsi = 0;
	Psia = 0;
	M2Psis = 0;
	M2Psit = 0;
	PlanPsi = 0;
}

void cTEMIm::freeMemory()
{
	GPU_Device = 0;
	MEffect = 0;
	Psirh = 0;
	Psiih = 0;
	E0 = 0;
	lx = 0;
	ly = 0;
	nx = 0;
	ny = 0;
	lambda = 0;

	cudaFreen(fPsi);
	cudaFreen(Psia);
	cudaFreen(M2Psis);
	cudaFreen(M2Psit);
	cufftDestroyn(PlanPsi);

	MT_MicroscopeEffects_GPU.freeMemory();
}

void cTEMIm::GenerateParameters()
{
	GP.nx = nx;
	GP.ny = ny;
	GP.nxh = GP.nx/2;
	GP.nyh = GP.ny/2;
	GP.nxy = GP.nx*GP.ny;
	GP.inxy = (GP.nxy==0)?(0.0):(1.0/GP.nxy);
	GP.dRx = (nx==0)?(0.0):(lx/GP.nx);
	GP.dRy = (ny==0)?(0.0):(ly/GP.ny);
	GP.dgx = (lx==0)?(0.0):(1.0/lx);
	GP.dgy = (ly==0)?(0.0):(1.0/ly);

	GP.gmax = MIN(GP.nxh*GP.dgx, GP.nyh*GP.dgy);
	GP.gmax2 = GP.gmax*GP.gmax;
	GP.gmaxl = 2.0*GP.gmax/3.0;
	GP.gmaxl2 = GP.gmaxl*GP.gmaxl;

	lambda = f_getLambda(E0);

	f_sLens_Cal(lambda, GP, Lens);	// Lens coefficients
	//f_sBT_Cal(GP, BT);			// Blocks and threads

	cudaFreen(fPsi);
	hipMalloc((void**)&fPsi, GP.nxy*cSizeofCD);
	cudaFreen(Psia);
	hipMalloc((void**)&Psia, GP.nxy*cSizeofCD);

	cudaFreen(M2Psis);
	hipMalloc((void**)&M2Psis, GP.nxy*cSizeofRD);
	cudaFreen(M2Psit);
	hipMalloc((void**)&M2Psit, GP.nxy*cSizeofRD);

	cufftDestroyn(PlanPsi); 
	hipfftPlan2d(&PlanPsi, nx, ny, HIPFFT_Z2Z);

	// Microscope parameters
	//MT_MicroscopeEffects_GPU.SetInputData(BT, GP, Lens, PlanPsi, fPsi, Psia, M2Psit);
}

void cTEMIm::SetInputData(sInTEMIm &InTEMIm)
{
	GPU_Device = InTEMIm.GPU_Device; 
	MEffect = InTEMIm.MEffect;
	Psirh= InTEMIm.Psirh;
	Psiih = InTEMIm.Psiih;
	E0 = InTEMIm.E0;
	lx = InTEMIm.lx;
	ly = InTEMIm.ly;
	nx = InTEMIm.nx;
	ny = InTEMIm.ny;
	Lens.m = InTEMIm.MC_m;
	Lens.f = InTEMIm.MC_f;
	Lens.Cs3 = InTEMIm.MC_Cs3;
	Lens.Cs5 = InTEMIm.MC_Cs5;
	Lens.mfa2 = InTEMIm.MC_mfa2;
	Lens.afa2 = InTEMIm.MC_afa2;
	Lens.mfa3 = InTEMIm.MC_mfa3;
	Lens.afa3 = InTEMIm.MC_afa3;
	Lens.aobjl = InTEMIm.MC_aobjl;
	Lens.aobju = InTEMIm.MC_aobju;
	Lens.sf = InTEMIm.MC_sf;
	Lens.nsf = InTEMIm.MC_nsf;
	Lens.beta = InTEMIm.MC_beta;
	Lens.nbeta = InTEMIm.MC_nbeta;

	hipSetDevice(GPU_Device);
	hipDeviceReset();

	GenerateParameters();

 }

// Partially coherent transfer function and Transmission cross coefficient
void cTEMIm::TEMImage(double *Psir_hi, double *Psii_hi, double *M2Psi_ho)
{
	sComplex Psi;
	Psi.real = M2Psis, Psi.imag = M2Psit;
	// Copy real part of Psi
	hipMemcpy(Psi.real, Psir_hi, GP.nxy*cSizeofRD, hipMemcpyHostToDevice);
	// Copy imaginary part of Psi
	hipMemcpy(Psi.imag, Psii_hi, GP.nxy*cSizeofRD, hipMemcpyHostToDevice);
	// Set real and imaginary part to Psi
	//f_Set_MC_GPU(BT.Bnxy, BT.Tnxy, GP.nxy, Psi, fPsi);
	// fft2shift
	//f_fft2Shift_MC_GPU(BT.Bhnxny, BT.Thnxny, GP.nxh, GP.nyh, fPsi);
	// Forward fft2
	hipfftExecZ2Z(PlanPsi, fPsi, fPsi, HIPFFT_FORWARD);

	/*********************Microscope effects**********************/
	//if(MEffect==0)
	//	MT_MicroscopeEffects_GPU.PCLIMWPOTEM(M2Psis);	
	//else
	//	MT_MicroscopeEffects_GPU.PCTCCTEM(M2Psis);
	/************************************************************/
	// fft2shift
	//f_fft2Shift_MD_GPU(BT.Bhnxny, BT.Thnxny, GP.nxh, GP.nyh, M2Psis);
	// copy M2Psi to the host
	hipMemcpy(M2Psi_ho, M2Psis, GP.nxy*cSizeofRD, hipMemcpyDeviceToHost);
}