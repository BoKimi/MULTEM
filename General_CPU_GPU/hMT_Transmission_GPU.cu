#include "hip/hip_runtime.h"
/*
 * This file is part of MULTEM.
 * Copyright 2014 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http://www.gnu.org/licenses/>.
 */

#include "math.h"

#include "hConstTypes.h"
#include "hQuadrature.h"
#include "hMT_General_CPU.h"
#include "hMT_General_GPU.h"
#include "hMT_MGP_CPU.h"
#include "hMT_AtomTypes_GPU.h"
#include "hMT_Transmission_GPU.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hipfft/hipfft.h>

// From double to float potential
__global__ void k_V0_D2F(sGP GP, const double * __restrict V0_i, float * __restrict Ve_o){
	int iy = threadIdx.x + blockIdx.x*blockDim.x;
	int ix = threadIdx.y + blockIdx.y*blockDim.y;

	if ((ix < GP.nx)&&(iy < GP.ny)){
		int ixy = ix*GP.ny+iy;
		double V0 = V0_i[ixy];
		Ve_o[ixy] = V0;
	}
}

// Calculated transmission function
template <class Type>
__global__ void k_Transmission(sGP GP, int ApproxModel, double f, const Type * __restrict V0_i, double2 * __restrict Trans_o){
	int iy = threadIdx.x + blockIdx.x*blockDim.x;
	int ix = threadIdx.y + blockIdx.y*blockDim.y;

	if ((ix < GP.nx)&&(iy < GP.ny)){
		int ixy = ix*GP.ny+iy;
		double V0 = V0_i[ixy];
		double theta = f*V0, x = 1.0, y = theta;
		if(ApproxModel!=4) sincos(theta, &y , &x);
		Trans_o[ixy].x = x;
		Trans_o[ixy].y = y;
	}
}

// Element by element multiplication
__global__ void k_Transmit(sGP GP, const double2 * __restrict Trans_i, double2 * __restrict Psi_io){
	int iy = threadIdx.x + blockIdx.x*blockDim.x;
	int ix = threadIdx.y + blockIdx.y*blockDim.y;

	if ((ix < GP.nx)&&(iy < GP.ny)){
		int ixy = ix*GP.ny+iy;
		double z1r = Trans_i[ixy].x, z1i = Trans_i[ixy].y;
		double z2r = Psi_io[ixy].x, z2i = Psi_io[ixy].y;
		double z3r = z1r*z2r-z1i*z2i, z3i = z1i*z2r+z1r*z2i;
		Psi_io[ixy].x = z3r;
		Psi_io[ixy].y = z3i;
	}
}

/***************************************************************************/
/***************************************************************************/

void cMT_Transmission_GPU::freeMemory(){
	hipDeviceSynchronize(); // wait to finish the work in the GPU

	cSynCPU = ccSynCPU;
	fPot = 0.0;

	cudaFreen(Trans0);

	if(nSliceMem0>0)
		if(SliceMemTyp==1){
			for(int iSliceMem=0; iSliceMem<nSliceMem0; iSliceMem++)
				cudaFreen(Trans[iSliceMem]);
			delete [] Trans; Trans = 0;
		}else{
			for(int iSliceMem=0; iSliceMem<nSliceMem0; iSliceMem++)
				cudaFreen(Vpe[iSliceMem]);
			delete [] Vpe; Vpe = 0;
		}

	SliceMemTyp = 0;
	nSliceMem = 0;
	nSliceMem0 = 0;

	PlanTrans = 0;
}

void cMT_Transmission_GPU::freeMemoryReset(){
	freeMemory();
	hipDeviceReset();
}

cMT_Transmission_GPU::cMT_Transmission_GPU()
{
	cSynCPU = ccSynCPU;
	fPot = 0.0;

	SliceMemTyp = 0;
	nSliceMem = 0;
	nSliceMem0 = 0;

	Trans0 = 0;
	Trans = 0;
	Vpe = 0;

	PlanTrans = 0;
}

cMT_Transmission_GPU::~cMT_Transmission_GPU(){
	freeMemory();
}

void cMT_Transmission_GPU::Cal_Trans_or_Vpe(){
	dim3 Bnxny, Tnxny;
	f_get_BTnxny(GP, Bnxny, Tnxny);

	for (int iSliceMem=0; iSliceMem<nSliceMem; iSliceMem++){
		ProjectedPotential(iSliceMem);
		if(SliceMemTyp==1){
			k_Transmission<double><<<Bnxny, Tnxny>>>(GP, MT_MGP_CPU->ApproxModel, fPot, V0, Trans[iSliceMem]);
			f_BandwidthLimit2D(PlanTrans, GP, Trans[iSliceMem]);
		}else
			k_V0_D2F<<<Bnxny, Tnxny>>>(GP, V0, Vpe[iSliceMem]);
	}
	hipDeviceSynchronize();
}

void cMT_Transmission_GPU::SetInputData(cMT_MGP_CPU *MT_MGP_CPU_io, hipfftHandle &PlanTrans_i, int nAtomsM_i, double *AtomsM_i){
	freeMemory();

	cMT_Potential_GPU::SetInputData(MT_MGP_CPU_io, nAtomsM_i, AtomsM_i);
	PlanTrans = PlanTrans_i;

	hipMalloc((void**)&Trans0, GP.nxy*cSizeofCD);

	fPot = f_getfPot(MT_MGP_CPU->E0, MT_MGP_CPU->theta);

	if((MT_MGP_CPU->FastCal==1)||(MT_MGP_CPU->ApproxModel>2)) return;

	int nSliceSigma = (MT_MGP_CPU->DimFP%10==0)?0:(int)ceil(6*sigma_max/MT_MGP_CPU->dz);
	int nSliceMax = nSlice + nSliceSigma;
	if(MT_MGP_CPU->MulOrder==2) nSliceMax++;

	size_t SizeFreeMem, SizeTotMem;
	hipMemGetInfo(&SizeFreeMem, &SizeTotMem);
	SizeFreeMem = SizeFreeMem-10*cMb;
	int nSliceMemMax = 0;

	if(SizeFreeMem/(GP.nxy*cSizeofCD)>=nSliceMax){
		SliceMemTyp = 1;
		nSliceMemMax = SizeFreeMem/(GP.nxy*cSizeofCD);
	}else{
		SliceMemTyp = 2;
		nSliceMemMax = SizeFreeMem/(GP.nxy*cSizeofRF);
	}

	if((nSliceMemMax>0)&&(MT_MGP_CPU->ApproxModel<=2)){
		nSliceMem0 = MIN(nSliceMemMax, nSliceMax);
		if(SliceMemTyp==1){
			Trans = new double2*[nSliceMem0];
			for(int iSliceMem=0; iSliceMem<nSliceMem0; iSliceMem++)
				hipMalloc((void**)&Trans[iSliceMem], GP.nxy*cSizeofCD);
		}else{
			Vpe = new float*[nSliceMem0];
			for(int iSliceMem=0; iSliceMem<nSliceMem0; iSliceMem++)
				hipMalloc((void**)&Vpe[iSliceMem], GP.nxy*cSizeofRF);
		}
	}

	nSliceMem = MIN(nSliceMem0, nSlice);
	if((MT_MGP_CPU->MulOrder==2)&&(nSliceMem0>nSlice)) nSliceMem++;
}

void cMT_Transmission_GPU::MoveAtoms(int iConf){
	cMT_Potential_GPU::MoveAtoms(iConf);
	int nSliceMem = MIN(nSliceMem0, nSlice);
	if((MT_MGP_CPU->MulOrder==2)&&(nSliceMem0>nSlice)) nSliceMem++;
	if(nSliceMem0>0) Cal_Trans_or_Vpe();

	if(MT_MGP_CPU->ApproxModel>2){
		dim3 Bnxny, Tnxny;
		f_get_BTnxny(GP, Bnxny, Tnxny);
		ProjectedPotential(0);
		k_Transmission<double><<<Bnxny, Tnxny>>>(GP, MT_MGP_CPU->ApproxModel, fPot, V0, Trans0);
		f_BandwidthLimit2D(PlanTrans, GP, Trans0);	
	}
}

double2* cMT_Transmission_GPU::getTrans(int iSlice, int typ){
	if(MT_MGP_CPU->ApproxModel>2) return Trans0;

	dim3 Bnxny, Tnxny;
	f_get_BTnxny(GP, Bnxny, Tnxny);

	double2 *Trans_o = Trans0;
	if(iSlice<nSliceMem){
		if(SliceMemTyp==1)
			Trans_o = Trans[iSlice];
		else{
			k_Transmission<float><<<Bnxny, Tnxny>>>(GP, MT_MGP_CPU->ApproxModel, fPot, Vpe[iSlice], Trans_o);
			f_BandwidthLimit2D(PlanTrans, GP, Trans_o);	
		}
	}else{
		ProjectedPotential(iSlice, typ);
		k_Transmission<double><<<Bnxny, Tnxny>>>(GP, MT_MGP_CPU->ApproxModel, fPot, V0, Trans_o);
		f_BandwidthLimit2D(PlanTrans, GP, Trans_o);	
	}
	return Trans_o;
}

void cMT_Transmission_GPU::Transmit(int iSlice, double2 *&Psi_io){
	dim3 Bnxny, Tnxny;
	f_get_BTnxny(GP, Bnxny, Tnxny);
	double2 *Trans = (MT_MGP_CPU->ApproxModel>2)?Trans0:getTrans(iSlice);
	k_Transmit<<<Bnxny, Tnxny>>>(GP, Trans, Psi_io);
}