#include "hip/hip_runtime.h"
/*
 * This file is part of MULTEM.
 * Copyright 2014 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstring>
#include "math.h"

#include "hConstTypes.h"
#include "hQuadrature.h"
#include "hMT_General_CPU.h"
#include "hMT_General_GPU.h"
#include "hMT_MGP_CPU.h"
#include "hMT_Specimen_CPU.h"
#include "hMT_AtomTypes_GPU.h"
#include "hMT_Potential_GPU.h"
#include "hMT_Transmission_GPU.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hipfft/hipfft.h>

// From double to float potential
__global__ void k_V0_D2F(sGP GP, const double * __restrict V0_i, float * __restrict Ve_o){
	int iy = threadIdx.x + blockIdx.x*blockDim.x;
	int ix = threadIdx.y + blockIdx.y*blockDim.y;

	if ((ix < GP.nx)&&(iy < GP.ny)){
		int ixy = ix*GP.ny+iy;
		double V0 = V0_i[ixy];
		Ve_o[ixy] = V0;
	}
}

// Calculated transmission function
template <class Type>
__global__ void k_Transmission(sGP GP, int ApproxModel, double f, const Type * __restrict V0_i, double2 * __restrict Trans_o){
	int iy = threadIdx.x + blockIdx.x*blockDim.x;
	int ix = threadIdx.y + blockIdx.y*blockDim.y;

	if ((ix < GP.nx)&&(iy < GP.ny)){
		int ixy = ix*GP.ny+iy;
		double V0 = V0_i[ixy];
		double theta = f*V0, x = 1.0, y = theta;
		if(ApproxModel~=4) sincos(theta, &y , &x);
		Trans_o[ixy].x = x;
		Trans_o[ixy].y = y;
	}
}

/***************************************************************************/
/***************************************************************************/

void cMT_Transmission_GPU::freeMemory(){
	hipDeviceSynchronize(); // wait to finish the work in the GPU

	cSynCPU = ccSynCPU;
	fPot = 0.0;

	cudaFreen(Trans0);

	if(nSliceM>0)
		if(SliceMTyp==1){
			for(int iSliceM=0; iSliceM<nSliceM; iSliceM++)
				cudaFreen(Trans[iSliceM]);
			delete [] Trans; Trans = 0;
		}else{
			for(int iSliceM=0; iSliceM<nSliceM; iSliceM++)
				cudaFreen(Vpe[iSliceM]);
			delete [] Vpe; Vpe = 0;
		}
	nSliceM = 0;
	SliceMTyp = 0;

	PlanTrans = 0;
}

cMT_Transmission_GPU::cMT_Transmission_GPU()
{
	cSynCPU = ccSynCPU;
	fPot = 0.0;

	nSliceM = 0;
	SliceMTyp = 0;

	Trans0 = 0;
	Trans = 0;
	Vpe = 0;

	PlanTrans = 0;
}

cMT_Transmission_GPU::~cMT_Transmission_GPU(){
	freeMemory();
}

void cMT_Transmission_GPU::f_V0_D2F(sGP &GP, double *&V0_i, float *&Ve_o){
	dim3 Bnxny, Tnxny;
	f_get_BTnxny(GP, Bnxny, Tnxny);	
	k_V0_D2F<<<Bnxny, Tnxny>>>(GP, V0_i, Trans_o);
}

double2* cMT_Transmission_GPU::Transmission(int iSlice){
	dim3 Bnxny, Tnxny;
	f_get_BTnxny(GP, Bnxny, Tnxny);

	double2 *Trans_o = Trans0;
	if(iSlice<nSliceM){
		if(SliceMTyp==1) 
			Trans_o = Trans[iSlice];
		else
			k_Transmission<float><<<Bnxny, Tnxny>>>(GP, ApproxModel, fPot, Vpe[iSlice], Trans_o);
	}else{
		ProjectedPotential(iSlice);
		k_Transmission<double><<<Bnxny, Tnxny>>>(GP, ApproxModel, fPot, V0, Trans_o);
	}
	f_BandwidthLimit2D(PlanTrans, GP, Trans_o);		// AntiAliasing
}

void cMT_Transmission_GPU::Transmission(double fPot, float *&V0_i, double2 *&Trans_o){
	dim3 Bnxny, Tnxny;
	f_get_BTnxny(GP, Bnxny, Tnxny);	
	k_Transmission<float><<<Bnxny, Tnxny>>>(GP, ApproxModel, fPot, V0_i, Trans_o);		// Transmission
	f_BandwidthLimit2D(PlanTrans, GP, Trans_o);											// AntiAliasing
}

void cMT_Transmission_GPU::Transmission(int iSlice, double fPot, double2 *&Trans){
	ProjectedPotential(iSlice); // Projected potential

	Transmission(PlanTrans, GP, MT_MGP_CPU->ApproxModel, fPot, V0, Trans);
}

void cMT_Transmission_GPU::Cal_Trans_Vpe(){
	int iSliceM, nSliceMm = MIN(nSliceM, nSlice);
	if((MT_MGP_CPU->MulOrder==2)&&(nSliceM>nSlice)) nSliceMm++;

	for (iSliceM=0; iSliceM<nSliceMm; iSliceM++){
		if(SliceMTyp==1)
			MT_MulSli_GPU->Transmission(iSliceM, Trans[iSliceM]);
		else
			Potential_Efective(iSliceM, MT_MulSli_GPU->fPot, Vpe[iSliceM]);
	}
	hipDeviceSynchronize();
}

void cMT_Transmission_GPU::Transmission_Transmit(int iSlice, double2 *&Psi){
	int nSlice = MT_Specimen_CPU->nSlice;
	int nSliceMm = MIN(nSliceM, nSlice);
	if((MT_MGP_CPU->MulOrder==2)&&(nSliceM>nSlice)) nSliceMm++;

	double2 *Transt;
	Transt = (SliceMTyp==1)?Trans[iSlice]:MT_MulSli_GPU->Trans;

	if(iSlice<nSliceMm){
		if(SliceMTyp==2)
			f_Transmission_1_2(MT_MulSli_GPU->PlanTrans, GP, Vpe[iSlice], Transt);
		MT_MulSli_GPU->Transmit(Transt, Psi);
	}else
		MT_MulSli_GPU->Transmission_Transmit(iSlice, Psi);
}

void cMT_Transmission_GPU::SetInputData(cMT_MGP_CPU *MT_MGP_CPU_io, int nAtomsM_i, double *AtomsM_i)
{
	freeMemory();

	cMT_Potential_GPU::SetInputData(MT_MGP_CPU_io, nAtomsM_i, AtomsM_i);

	double Gamma = f_getGamma(MT_MGP_CPU->E0);
	double Lambda = f_getLambda(MT_MGP_CPU->E0);
	fPot = Gamma*Lambda/(cPotf*cos(MT_MGP_CPU->theta));

	nSliceM = MIN(nSliceM0, nSlice);
	if((MT_MGP_CPU->MulOrder==2)&&(nSliceM0>nSlice)) nSliceM++;

	int nSliceSigma = ((MT_MGP_CPU->ApproxModel>1)||(MT_MGP_CPU->DimFP%10==0))?0:(int)ceil(6*sigma_max/MT_MGP_CPU->dz);
	int nSliceMax = nSlice + nSliceSigma;
	nSliceMax = (MT_MGP_CPU->MulOrder==1)?nSliceMax:nSliceMax+1;

	size_t SizeFreeM, SizeTotM;
	hipMemGetInfo(&SizeFreeM, &SizeTotM);
	SizeFreeM = SizeFreeM-10*cMb;
	int nSliceMt;

	if(SizeFreeM/(GP.nxy*cSizeofCD)>=nSliceMax){
		SliceMTyp = 1;
		nSliceMt = SizeFreeM/(GP.nxy*cSizeofCD);
	}else{
		SliceMTyp = 2;
		nSliceMt = SizeFreeM/(GP.nxy*cSizeofRF);
	}

	if((nSliceMt>0)&&(MT_MGP_CPU->SimType==1)&&(MT_MGP_CPU->ApproxModel<=2)){
		nSliceM = MIN(nSliceMt, nSliceMax);
		if(SliceMTyp==1){
			Trans = new double2*[nSliceM];
			for(int iSliceM=0; iSliceM<nSliceM; iSliceM++)
				hipMalloc((void**)&Trans[iSliceM], GP.nxy*cSizeofCD);
		}else{
			Vpe = new float*[nSliceM];
			for(int iSliceM=0; iSliceM<nSliceM; iSliceM++)
				hipMalloc((void**)&Vpe[iSliceM], GP.nxy*cSizeofRF);
		}
	}
}