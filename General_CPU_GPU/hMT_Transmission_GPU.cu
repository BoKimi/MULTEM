#include "hip/hip_runtime.h"
/*
 * This file is part of MULTEM.
 * Copyright 2014 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstring>
#include "math.h"

#include "hConstTypes.h"
#include "hQuadrature.h"
#include "hMT_General_CPU.h"
#include "hMT_General_GPU.h"
#include "hMT_MGP_CPU.h"
#include "hMT_Specimen_CPU.h"
#include "hMT_AtomTypes_GPU.h"
#include "hMT_Potential_GPU.h"
#include "hMT_Transmission_GPU.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hipfft/hipfft.h>

void cMT_Transmission_GPU::freeMemory(){
	hipDeviceSynchronize(); // wait to finish the work in the GPU

	cSynCPU = ccSynCPU;

	if(nSliceM>0)
		if(SliceMTyp==1){
			for(int iSliceM=0; iSliceM<nSliceM; iSliceM++)
				cudaFreen(Trans[iSliceM]);
			delete [] Trans; Trans = 0;
		}else{
			for(int iSliceM=0; iSliceM<nSliceM; iSliceM++)
				cudaFreen(Vpe[iSliceM]);
			delete [] Vpe; Vpe = 0;
		}

	nSliceM = 0;
	SliceMTyp = 0;
}

cMT_Transmission_GPU::cMT_Transmission_GPU()
{
	cSynCPU = ccSynCPU;

	nSliceM = 0;
	SliceMTyp = 0;

	Trans = 0;
	Vpe = 0;
}

cMT_Transmission_GPU::~cMT_Transmission_GPU(){
	freeMemory();
}

void cMT_Transmission_GPU::Potential_Efective(int iSlice, double fPot, float *&Vpe){
	eSlicePos SlicePos = (iSlice==0)?eSPFirst:(iSlice<nSlice)?eSPMedium:eSPLast;

	// Projected potential
	if(iSlice<nSlice) ProjectedPotential(iSlice);

	switch (MT_MGP_CPU.MulOrder)
	{
		case 1:
			f_Potential1(GP, fPot, V0, Vpe);
			break;
		case 2:
			f_Potential2(GP, fPot, V0, V1, V2, SlicePos, Vpe);
			break;
	}
}

void cMT_Transmission_GPU::Transmission(int iSlice, double2 *&Trans){
	eSlicePos SlicePos = (iSlice==0)?eSPFirst:(iSlice<nSlice)?eSPMedium:eSPLast;

	// Projected potential
	if(iSlice<nSlice) ProjectedPotential(iSlice); 

	switch (MT_MGP_CPU.MulOrder)
	{
		case 1:
			if(MT_MGP_CPU.ApproxModel==4) 
				f_TransmissionWPO(PlanPsi, GP, fPot, V0, Trans);
			else 
				f_Transmission1(PlanPsi, GP, fPot, V0, Trans);
			break;
		case 2:
			f_Transmission2(PlanPsi, GP, fPot, MT_Potential_GPU->V0, MT_Potential_GPU->V1, MT_Potential_GPU->V2, SlicePos, Trans);
			break;
	}
}

void cMT_Transmission_GPU::Cal_Trans_Vpe(){
	int iSliceM, nSliceMm = MIN(nSliceM, nSlice);
	if((MT_MGP_CPU.MulOrder==2)&&(nSliceM>nSlice)) nSliceMm++;

	for (iSliceM=0; iSliceM<nSliceMm; iSliceM++){
		if(SliceMTyp==1)
			MT_MulSli_GPU->Transmission(iSliceM, Trans[iSliceM]);
		else
			Potential_Efective(iSliceM, MT_MulSli_GPU->fPot, Vpe[iSliceM]);
	}
	hipDeviceSynchronize();
}

void cMT_Transmission_GPU::Transmission_Transmit(int iSlice, double2 *&Psi){
	int nSlice = MT_Specimen_CPU->nSlice;
	int nSliceMm = MIN(nSliceM, nSlice);
	if((MT_MGP_CPU.MulOrder==2)&&(nSliceM>nSlice)) nSliceMm++;

	double2 *Transt;
	Transt = (SliceMTyp==1)?Trans[iSlice]:MT_MulSli_GPU->Trans;

	if(iSlice<nSliceMm){
		if(SliceMTyp==2)
			f_Transmission_1_2(MT_MulSli_GPU->PlanPsi, GP, Vpe[iSlice], Transt);
		MT_MulSli_GPU->Transmit(Transt, Psi);
	}else
		MT_MulSli_GPU->Transmission_Transmit(iSlice, Psi);
}

void cMT_Transmission_GPU::Cal_FAST_STEM_Wavefunction_POA_WPOA(int nConfFP, sDetInt *DetInt){
	int iSlice = 0;
	int ist, iThk = 0;
	int iConf0 = (nConfFP==0)?0:1;
	double inConfFP = (nConfFP==0)?1.0:1.0/double(nConfFP);
	double nxy2 = pow(double(GP.nxy), 2);

	InitImSTEM();
	for (int iConf=iConf0; iConf<=nConfFP; iConf++){
		// Move atoms
		MT_Specimen_CPU->MoveAtoms(iConf);
		// Transmission
		MT_MulSli_GPU->Transmission(iSlice, MT_MulSli_GPU->Trans);
		for (ist=0; ist<nst; ist++){
			// Plane wave ilumination
			MT_IncidentWave_GPU->Psi0(xst[ist], yst[ist], MT_MulSli_GPU->Psi);
			// Transmit
			MT_MulSli_GPU->Transmit(MT_MulSli_GPU->Trans, MT_MulSli_GPU->Psi);
			// Inclined ilumination
			MT_MulSli_GPU->PhaseMul(MT_MulSli_GPU->Psi);
			// Backward fft2
			hipfftExecZ2Z(MT_MulSli_GPU->PlanPsi, MT_MulSli_GPU->Psi, MT_MulSli_GPU->Psi, HIPFFT_FORWARD);
			// Add Psi to aM2Psi
			f_Add_wMC2(false, GP, inConfFP/nxy2, MT_MulSli_GPU->Psi, MT_MulSli_GPU->aM2Psi);
			// Detector integration
			MT_Detector_GPU->getDetectorIntensity(MT_MulSli_GPU->aM2Psi, ist, ImSTEM[iThk].DetInt, true);
		}
	}
}

void cMT_Transmission_GPU::Cal_FAST_STEM_Wavefunction_MSA(int nConfFP, sDetInt *DetInt){
	int iSlice = 0, iSynCPU = 0;
	int ist, iThk = 0;
	int iConf0 = (nConfFP==0)?0:1;
	double inConfFP = (nConfFP==0)?1.0:1.0/double(nConfFP);
	double nxy2 = pow(double(GP.nxy), 2);

	InitImSTEM();
	for (int iConf=iConf0; iConf<=nConfFP; iConf++){
		// Move atoms
		MT_Specimen_CPU->MoveAtoms(iConf);

		//Load Trans or Vpe
		Cal_Trans_Vpe();
		for (ist=0; ist<nst; ist++){
			// Plane wave ilumination
			MT_IncidentWave_GPU->Psi0(xst[ist], yst[ist], MT_MulSli_GPU->Psi);
			for (iSlice = 0; iSlice<MT_Specimen_CPU->nSlice; iSlice++){
				// Transmission and Transmit
				Transmission_Transmit(iSlice, MT_MulSli_GPU->Psi);
				// Propagate
				MT_MulSli_GPU->Propagate(eSReal, MT_MulSli_GPU->gxu, MT_MulSli_GPU->gyu, MT_Specimen_CPU->get_dz(iSlice), MT_MulSli_GPU->Psi);
				// GPU Synchronize
				f_GPU_Sync_CPU(iSynCPU, cSynCPU); 
			}
			// Last Transmission and Transmit
			if (MT_MGP_CPU.MulOrder==2) Transmission_Transmit(iSlice, MT_MulSli_GPU->Psi);
			// Inclined ilumination
			MT_MulSli_GPU->PhaseMul(MT_MulSli_GPU->Psi);
			// Backward fft2
			hipfftExecZ2Z(MT_MulSli_GPU->PlanPsi, MT_MulSli_GPU->Psi, MT_MulSli_GPU->Psi, HIPFFT_FORWARD);
			// Add Psi to aM2Psi
			f_Add_wMC2(false, GP, inConfFP/nxy2, MT_MulSli_GPU->Psi, MT_MulSli_GPU->aM2Psi);
			// Detector integration
			MT_Detector_GPU->getDetectorIntensity(MT_MulSli_GPU->aM2Psi, ist, ImSTEM[iThk].DetInt, true);
		}
	}
}

void cMT_Transmission_GPU::SetInputData(cMT_MGP_CPU &MT_MGP_CPU_io, int nAtomsM_i, double *AtomsM_i)
{
	freeMemory();

	cMT_Potential_GPU::SetInputData(MT_MGP_CPU_io, nAtomsM_i, AtomsM_i, GP.dRmin);
	MT_MGP_CPU_io = MT_MGP_CPU;

	double gamma = f_getGamma(MT_MGP_CPU.E0);

	doublelambda = f_getLambda(E0);

	fPot = Lens.gamma*Lens.lambda/(cPotf*cos(MT_MGP_CPU.theta));

	MT_MulSli_GPU = MT_MulSli_GPU_i;
	MT_Potential_GPU = MT_MulSli_GPU->MT_Potential_GPU;
	MT_Specimen_CPU = MT_Potential_GPU->MT_Specimen_CPU;
	MT_IncidentWave_GPU = MT_MulSli_GPU->MT_IncidentWave_GPU;
	
	MT_MGP_CPU = MT_MulSli_GPU->MT_MGP_CPU;
	GP = MT_MulSli_GPU->GP;

	line = MT_InMulSli_CPU.STEM_line;
	FastCal = MT_InMulSli_CPU.STEM_FastCal;
	ns = MT_InMulSli_CPU.STEM_ns;
	x1u = MT_InMulSli_CPU.STEM_x1u;	
	y1u = MT_InMulSli_CPU.STEM_y1u;
	x2u = MT_InMulSli_CPU.STEM_x2u;
	y2u = MT_InMulSli_CPU.STEM_y2u;
	f_BuildGrid(line, ns, x1u, y1u, x2u, y2u, nxs, nys, xs, ys);

	nThk = MT_MGP_CPU.nThk;
	if(nThk>0){
		Thk = new double[nThk];
		memcpy(Thk, MT_InMulSli_CPU.Thickness, nThk*cSizeofRD);
	}

	nDet = MT_InMulSli_CPU.STEM_nDet;
	double lambda = f_getLambda(MT_InMulSli_CPU.E0);
	f_sDetCir_Malloc(nDet, DetCir);
	for (int iDet=0; iDet<nDet; iDet++){
		DetCir.g2min[iDet] = pow(MT_InMulSli_CPU.STEM_DetCir[iDet].InnerAng/lambda, 2);
		DetCir.g2max[iDet] = pow(MT_InMulSli_CPU.STEM_DetCir[iDet].OuterAng/lambda, 2);
	}

	MT_Detector_GPU = new cMT_Detector_GPU;
	MT_Detector_GPU->SetInputData(GP, nDet, DetCir);

	nst = (line==1)?ns:nxs*nys;
	int ils, ixs, iys, ixys;
	xst = new double[nst];
	yst = new double[nst];
	if(line==1){
		for (ils=0; ils<ns; ils++){
			xst[ils] = xs[ils];
			yst[ils] = ys[ils];
		}
	}else{
		for (ixs=0; ixs<nxs; ixs++)
			for (iys=0; iys<nys; iys++){
				ixys = ixs*nys + iys;
				xst[ixys] = xs[ixs];
				yst[ixys] = ys[iys];
			}
	}

	int iThk, iDet, ist;
	ImSTEM = new sImSTEM[nThk];
	for (iThk = 0; iThk<nThk; iThk++){
		ImSTEM[iThk].DetInt = new sDetInt[nDet];
		for (iDet=0; iDet<nDet; iDet++){
			ImSTEM[iThk].DetInt[iDet].Coh = new double[nst];
			ImSTEM[iThk].DetInt[iDet].Tot = new double[nst];
			for (ist=0; ist<nst; ist++){
				ImSTEM[iThk].DetInt[iDet].Coh[ist] = 0.0;
				ImSTEM[iThk].DetInt[iDet].Tot[ist] = 0.0;
			}
		}
	}

	/****************************************************************/
	int nSliceSigma = ((MT_MGP_CPU.ApproxModel>1)||(MT_MGP_CPU.DimFP%10==0))?0:(int)ceil(6*MT_Specimen_CPU->sigma_max/MT_MGP_CPU.dz);
	int nSliceMax = MT_Specimen_CPU->nSlice + nSliceSigma;
	nSliceMax = (MT_MGP_CPU.MulOrder==1)?nSliceMax:nSliceMax+1;

	size_t SizeFreeM, SizeTotM;
	hipMemGetInfo(&SizeFreeM, &SizeTotM);
	SizeFreeM = SizeFreeM-10*cMb;
	int nSliceMt;

	if(SizeFreeM/(GP.nxy*cSizeofCD)>=nSliceMax){
		SliceMTyp = 1;
		nSliceMt = SizeFreeM/(GP.nxy*cSizeofCD);
	}else{
		SliceMTyp = 2;
		nSliceMt = SizeFreeM/(GP.nxy*cSizeofRF);
	}

	if((FastCal)&&(nSliceMt>0)&&(MT_MGP_CPU.SimType==1)&&(MT_MGP_CPU.ApproxModel<=2)){
		nSliceM = MIN(nSliceMt, nSliceMax);
		if(SliceMTyp==1){
			Trans = new double2*[nSliceM];
			for(int iSliceM=0; iSliceM<nSliceM; iSliceM++)
				hipMalloc((void**)&Trans[iSliceM], GP.nxy*cSizeofCD);
		}else{
			Vpe = new float*[nSliceM];
			for(int iSliceM=0; iSliceM<nSliceM; iSliceM++)
				hipMalloc((void**)&Vpe[iSliceM], GP.nxy*cSizeofRF);
		}
	}
}
