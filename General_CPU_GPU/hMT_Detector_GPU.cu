#include "hip/hip_runtime.h"
/*
 * This file is part of MULTEM.
 * Copyright 2014 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http://www.gnu.org/licenses/>.
 */

#include "math.h"
#include <cstring>
#include "hConstTypes.h"
#include "hMT_General_CPU.h"
#include "hMT_General_GPU.h"
#include "hMT_Detector_GPU.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

void cMT_Detector_GPU::freeMemory()
{
	if(IdCall==0) return;

	hipDeviceSynchronize(); // wait to finish the work in the GPU

	f_sGP_Init(GP);

	nDet = 0;
	f_sDetCir_Free_CPU(DetCirh);

	delete [] Tot_h; Tot_h = 0;
	delete [] Coh_h; Coh_h = 0;

	cudaFreen(Tot_d);
	cudaFreen(Coh_d);

	cudaFreen(M1p_d);
	cudaFreen(M2p_d);
}

cMT_Detector_GPU::cMT_Detector_GPU()
{
	IdCall = 0;

	f_sGP_Init(GP);

	nDet = 0;
	f_sDetCir_Init_CPU(DetCirh);

	Tot_h = 0;
	Coh_h = 0;

	Tot_d = 0;
	Coh_d = 0;

	M1p_d = 0;
	M2p_d = 0;
}

cMT_Detector_GPU::~cMT_Detector_GPU()
{
	freeMemory();
	IdCall = 0;
}

void cMT_Detector_GPU::SetInputData(sGP &GP_i, int nDeti, sDetCir &DetCirhi)
{
	freeMemory();
	IdCall++;

	GP = GP_i;

	nDet = nDeti;
	f_sDetCir_Malloc_CPU(nDet, DetCirh);
	memcpy(DetCirh.g2min, DetCirhi.g2min, nDet*cSizeofRD);
	memcpy(DetCirh.g2max, DetCirhi.g2max, nDet*cSizeofRD);

	Tot_h = new double[nDet];
	Coh_h = new double[nDet];

	hipMalloc((void**)&Tot_d, nDet*cSizeofRD);
	hipMalloc((void**)&Coh_d, nDet*cSizeofRD);

	hipMalloc((void**)&M1p_d, 32*32*cSizeofRD);
	hipMalloc((void**)&M2p_d, 32*32*cSizeofRD);
}

void cMT_Detector_GPU::getDetectorIntensity(double w_i, double *&aM2Psi, double *&M2aPsi, int ixys, sDetInt *DetInth)
{
	for(int iDet = 0; iDet<nDet; iDet++)
		f_Sum_MD_Det_GPU(GP, w_i, aM2Psi, M2aPsi, DetCirh.g2min[iDet], DetCirh.g2max[iDet], M1p_d, M2p_d, iDet, Tot_d, Coh_d);

	hipMemcpy(Tot_h, Tot_d, nDet*cSizeofRD, hipMemcpyDeviceToHost);
	hipMemcpy(Coh_h, Coh_d, nDet*cSizeofRD, hipMemcpyDeviceToHost);

	for(int iDet = 0; iDet<nDet; iDet++)
	{
		DetInth[iDet].Tot[ixys] = Tot_h[iDet];
		DetInth[iDet].Coh[ixys] = Coh_h[iDet];
	}
}

void cMT_Detector_GPU::getDetectorIntensity(double w_i, double *&aM2Psi, int ixys, sDetInt *DetInth, bool add)
{
	for(int iDet = 0; iDet<nDet; iDet++)
		f_Sum_MD_Det_GPU(GP, w_i, aM2Psi, DetCirh.g2min[iDet], DetCirh.g2max[iDet], M1p_d, iDet, Tot_d);

	hipMemcpy(Tot_h, Tot_d, nDet*cSizeofRD, hipMemcpyDeviceToHost);

	for(int iDet = 0; iDet<nDet; iDet++)
	{
		if(add)
		{
			DetInth[iDet].Tot[ixys] += Tot_h[iDet];
		}
		else
		{
			DetInth[iDet].Tot[ixys] = Tot_h[iDet];
		}
	}
}

void cMT_Detector_GPU::getDetectorIntensity(double w_i, double2 *&aPsi, int ixys, sDetInt *DetInth, bool add)
{
	for(int iDet = 0; iDet<nDet; iDet++)
		f_Sum_MC_Det_GPU(GP, w_i, aPsi, DetCirh.g2min[iDet], DetCirh.g2max[iDet], M1p_d, iDet, Tot_d);

	hipMemcpy(Tot_h, Tot_d, nDet*cSizeofRD, hipMemcpyDeviceToHost);

	for(int iDet = 0; iDet<nDet; iDet++)
	{
		if(add)
		{
			DetInth[iDet].Tot[ixys] += Tot_h[iDet];
		}
		else
		{
			DetInth[iDet].Tot[ixys] = Tot_h[iDet];
		}
	}
}
