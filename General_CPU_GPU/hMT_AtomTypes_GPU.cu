/*
 * This file is part of MULTEM.
 * Copyright 2014 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http://www.gnu.org/licenses/>.
 */

#include "hConstTypes.h"
#include "hMT_General_GPU.h"
#include "hMT_AtomTypes_CPU.h"
#include "hMT_AtomTypes_GPU.h"

#include "math.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

// free memory
void cMT_AtomTypes_GPU::freeMemory(){
	hipDeviceSynchronize(); // wait to finish the work in the GPU

	Z = 0;
	m = 0;
	A = 0;
	rn_e = 0;
	rn_c = 0;
	ra_e = 0;
	ra_c = 0;
	Rmin = 0;
	Rmax = 0;
	Rmin2 = 0;
	Rmax2 = 0;

	f_sCoefPar_cudaFree(cfeg);
	f_sCoefPar_cudaFree(cfxg);
	f_sCoefPar_cudaFree(cPr);
	f_sCoefPar_cudaFree(cVr);
	f_sCoefPar_cudaFree(cVR);

	nR = 0;
	cudaFreen(R);
	cudaFreen(R2);
	f_sciVn_cudaFree(ciVR);
};

// Set Atom type
void cMT_AtomTypes_GPU::SetAtomTypes(cMT_AtomTypes_CPU &MT_AtomTypes_CPU_i){
	freeMemory(); // clean GPU memory

	Z = MT_AtomTypes_CPU_i.Z;
	m = MT_AtomTypes_CPU_i.m;
	A = MT_AtomTypes_CPU_i.A;
	rn_e = MT_AtomTypes_CPU_i.rn_e;
	rn_c = MT_AtomTypes_CPU_i.rn_c;
	ra_e = MT_AtomTypes_CPU_i.ra_e;
	ra_c = MT_AtomTypes_CPU_i.ra_c;
	Rmin = MT_AtomTypes_CPU_i.Rmin;
	Rmin2 = MT_AtomTypes_CPU_i.Rmin2;
	Rmax = MT_AtomTypes_CPU_i.Rmax;
	Rmax2 = MT_AtomTypes_CPU_i.Rmax2;

	f_sCoefPar_cudaMalloc(6, cfeg);
	hipMemcpy(cfeg.cl, MT_AtomTypes_CPU_i.cfeg.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cfeg.cnl, MT_AtomTypes_CPU_i.cfeg.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	f_sCoefPar_cudaMalloc(6, cfxg);
	hipMemcpy(cfxg.cl, MT_AtomTypes_CPU_i.cfxg.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cfxg.cnl, MT_AtomTypes_CPU_i.cfxg.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	f_sCoefPar_cudaMalloc(6, cPr);
	hipMemcpy(cPr.cl, MT_AtomTypes_CPU_i.cPr.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cPr.cnl, MT_AtomTypes_CPU_i.cPr.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	f_sCoefPar_cudaMalloc(6, cVr);
	hipMemcpy(cVr.cl, MT_AtomTypes_CPU_i.cVr.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cVr.cnl, MT_AtomTypes_CPU_i.cVr.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	f_sCoefPar_cudaMalloc(6, cVR);
	hipMemcpy(cVR.cl, MT_AtomTypes_CPU_i.cVR.cl, 6*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(cVR.cnl, MT_AtomTypes_CPU_i.cVR.cnl, 6*cSizeofRD, hipMemcpyHostToDevice);

	nR = MT_AtomTypes_CPU_i.nR;

	hipMalloc((void**)&R, nR*cSizeofRD);
	hipMemcpy(R, MT_AtomTypes_CPU_i.R, nR*cSizeofRD, hipMemcpyHostToDevice);

	hipMalloc((void**)&R2, nR*cSizeofRD);
	hipMemcpy(R2, MT_AtomTypes_CPU_i.R2, nR*cSizeofRD, hipMemcpyHostToDevice);

	f_sciVn_cudaMalloc(nR, ciVR);
	hipMemcpy(ciVR.c0, MT_AtomTypes_CPU_i.ciVR.c0, nR*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(ciVR.c1, MT_AtomTypes_CPU_i.ciVR.c1, nR*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(ciVR.c2, MT_AtomTypes_CPU_i.ciVR.c2, nR*cSizeofRD, hipMemcpyHostToDevice);
	hipMemcpy(ciVR.c3, MT_AtomTypes_CPU_i.ciVR.c3, nR*cSizeofRD, hipMemcpyHostToDevice);
}